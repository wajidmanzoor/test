#include "hip/hip_runtime.h"

#include "../inc/gpuMemoryAllocation.cuh"
#include "../utils/cuda_utils.cuh"

void memoryAllocationGraph(deviceGraphPointers &G, Graph &graph) {
  ui n = graph.n;
  ui m = graph.m;
  chkerr(hipMalloc((void **)&(G.offset), (n + 1) * sizeof(ui)));
  chkerr(hipMemcpy(G.offset, graph.offset.data(), (n + 1) * sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.neighbors), (2 * m) * sizeof(ui)));
  chkerr(hipMemcpy(G.neighbors, graph.neighbors.data(), (2 * m) * sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.degree), n * sizeof(ui)));
  chkerr(hipMemcpy(G.degree, graph.degree.data(), n * sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.cliqueDegree), n * sizeof(ui)));
  chkerr(hipMalloc((void **)&(G.cliqueCore), n * sizeof(int)));
  chkerr(hipMalloc((void **)&(G.cliqueCorePeelSequence), n * sizeof(ui)));
  chkerr(hipMalloc((void **)&(G.density), n * sizeof(double)));
  chkerr(hipMalloc((void **)&(G.motifCount), n * sizeof(ui)));

  hipDeviceSynchronize();
}

void memoryAllocationDAG(deviceDAGpointer &D, ui n, ui m) {
  chkerr(hipMalloc((void **)&(D.offset), (n + 1) * sizeof(ui)));
  chkerr(hipMemset(D.offset, 0, (n + 1) * sizeof(ui)));

  chkerr(hipMalloc((void **)&(D.neighbors), m * sizeof(ui)));
  chkerr(hipMalloc((void **)&(D.degree), n * sizeof(ui)));
  hipDeviceSynchronize();
}

void memoryAllocationComponent(deviceComponentPointers &C, ui n, ui m) {
  chkerr(hipMalloc((void **)&(C.componentOffset), (n + 1) * sizeof(ui)));
  chkerr(hipMemset(C.componentOffset, 0, (n + 1) * sizeof(ui)));
  chkerr(hipMalloc((void **)&(C.components), n * sizeof(ui)));
  chkerr(hipMalloc((void **)&(C.mapping), n * sizeof(ui)));
  chkerr(hipMalloc((void **)&(C.reverseMapping), n * sizeof(ui)));
  hipDeviceSynchronize();
}

void memoryAllocationTrie(deviceCliquesPointer &C, ui t, ui k) {
  chkerr(hipMalloc((void **)&(C.trie), (t * k) * sizeof(ui)));
  chkerr(hipMalloc((void **)&(C.status), t * sizeof(int)));
  hipDeviceSynchronize();
}

ui memoryAllocationlevelData(cliqueLevelDataPointer &L, ui k, ui pSize,
                             ui cpSize, ui maxDegree, ui totalWarps) {
  size_t partialSize = (size_t)totalWarps * pSize;
  size_t candidateSize = (size_t)totalWarps * cpSize;
  size_t offsetSize = (size_t)((pSize / (k - 1)) + 1) * totalWarps;
  ui maxBitMask = (maxDegree + 31) / 32;
  size_t maskSize = (size_t)cpSize * maxBitMask * totalWarps;
  ui max_ = partialSize / (k - 1);

  chkerr(hipMalloc((void **)&(L.partialCliquesPartition),
                    partialSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(L.partialCliques), partialSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(L.candidatesPartition),
                    candidateSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(L.candidates), candidateSize * sizeof(ui)));

  chkerr(
      hipMalloc((void **)&(L.validNeighMaskPartition), maskSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(L.validNeighMask), maskSize * sizeof(ui)));

  chkerr(hipMemset(L.validNeighMask, 0, maskSize * sizeof(ui)));
  chkerr(hipMemset(L.validNeighMaskPartition, 0, maskSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(L.offsetPartition), offsetSize * sizeof(ui)));
  chkerr(hipMemset(L.offsetPartition, 0, offsetSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(L.offset), offsetSize * sizeof(ui)));
  chkerr(hipMemset(L.offset, 0, offsetSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(L.count), (totalWarps + 1) * sizeof(ui)));
  chkerr(hipMalloc((void **)&(L.temp), (totalWarps + 1) * sizeof(ui)));
  chkerr(hipMemset(L.temp, 0, (totalWarps + 1) * sizeof(ui)));
  chkerr(hipMemset(L.count, 0, (totalWarps + 1) * sizeof(ui)));

  chkerr(hipMalloc((void **)&(L.max), sizeof(ui)));
  chkerr(hipMemcpy(L.max, &max_, sizeof(ui), hipMemcpyHostToDevice));

  hipDeviceSynchronize();
  return maxBitMask;
}

void memoryAllocationDensestCore(densestCorePointer &C, ui n, ui density,
                                 ui totalCliques, ui graphsize) {

  chkerr(hipMalloc((void **)&(C.mapping), n * sizeof(ui)));

  chkerr(hipMalloc((void **)&(C.offset), (n + 1) * sizeof(ui)));
  chkerr(hipMemset(C.offset, 0, (n + 1) * sizeof(ui)));

  // neighbors will be allocated once we now the size

  chkerr(hipMalloc((void **)&(C.cliqueDegree), n * sizeof(ui)));
  // chkerr(hipMalloc((void**)&(C.cliqueCore), n * sizeof(ui)));

  chkerr(hipMalloc((void **)&(C.density), n * sizeof(double)));
  chkerr(
      hipMemcpy(C.density, &density, sizeof(double), hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(C.n), sizeof(ui)));
  chkerr(hipMemcpy(C.n, &n, sizeof(ui), hipMemcpyHostToDevice));
  chkerr(hipMalloc((void **)&(C.m), sizeof(ui)));
  chkerr(hipMalloc((void **)&(C.totalCliques), sizeof(ui)));
  chkerr(hipMemcpy(C.totalCliques, &totalCliques, sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(C.reverseMap), graphsize * sizeof(ui)));
  hipDeviceSynchronize();
}

void memoryAllocationPrunnedNeighbors(devicePrunedNeighbors &prunedNeighbors,
                                      ui n, ui m) {
  chkerr(
      hipMalloc((void **)&(prunedNeighbors.newOffset), (n + 1) * sizeof(ui)));
  chkerr(hipMemset(prunedNeighbors.newOffset, 0, (n + 1) * sizeof(ui)));

  chkerr(hipMalloc((void **)&(prunedNeighbors.pruneStatus),
                    (2 * m) * sizeof(ui)));
  hipDeviceSynchronize();
}

void memoryAllocationFlowNetwork(deviceFlowNetworkPointers &flowNetwork,
                                 ui vertexSize, ui neighborSize) {

  chkerr(hipMalloc((void **)&(flowNetwork.height), (vertexSize) * sizeof(ui)));
  chkerr(hipMalloc((void **)&(flowNetwork.excess),
                    (vertexSize) * sizeof(double)));

  chkerr(hipMalloc((void **)&(flowNetwork.offset),
                    (vertexSize + 1) * sizeof(ui)));
  chkerr(
      hipMalloc((void **)&(flowNetwork.neighbors), neighborSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(flowNetwork.capacity),
                    neighborSize * sizeof(double)));
  chkerr(
      hipMalloc((void **)&(flowNetwork.flow), neighborSize * sizeof(double)));

  chkerr(
      hipMalloc((void **)&(flowNetwork.flowIndex), neighborSize * sizeof(ui)));

  hipDeviceSynchronize();
}

// Memory deallocation functions
void freeGraph(deviceGraphPointers &G) {
  chkerr(hipFree(G.offset));
  chkerr(hipFree(G.neighbors));
  chkerr(hipFree(G.degree));
  chkerr(hipFree(G.cliqueDegree));
  chkerr(hipFree(G.cliqueCore));
  chkerr(hipFree(G.cliqueCorePeelSequence));
  chkerr(hipFree(G.density));
  chkerr(hipFree(G.motifCount));
}

void freeComponents(deviceComponentPointers &C) {
  chkerr(hipFree(C.componentOffset));
  chkerr(hipFree(C.components));
  chkerr(hipFree(C.mapping));
}

void freeTrie(deviceCliquesPointer &C) {
  chkerr(hipFree(C.trie));
  chkerr(hipFree(C.status));
}

void freeDAG(deviceDAGpointer &D) {
  chkerr(hipFree(D.offset));
  chkerr(hipFree(D.neighbors));
  chkerr(hipFree(D.degree));
}

void freeLevelData(cliqueLevelDataPointer &L) {
  chkerr(hipFree(L.partialCliques));
  chkerr(hipFree(L.candidates));
  chkerr(hipFree(L.offset));
  chkerr(hipFree(L.validNeighMask));
  chkerr(hipFree(L.count));
  chkerr(hipFree(L.max));
  chkerr(hipFree(L.partialCliquesPartition));
  chkerr(hipFree(L.candidatesPartition));
  chkerr(hipFree(L.offsetPartition));
  chkerr(hipFree(L.validNeighMaskPartition));
  chkerr(hipFree(L.temp));
}

void freeDensestCore(densestCorePointer &C) {
  chkerr(hipFree(C.mapping));
  chkerr(hipFree(C.offset));
  chkerr(hipFree(C.neighbors));
  chkerr(hipFree(C.density));
  chkerr(hipFree(C.n));
  chkerr(hipFree(C.m));
  chkerr(hipFree(C.totalCliques));
  chkerr(hipFree(C.cliqueDegree));
  // chkerr(hipFree(C.cliqueCore));
}

void freePruneneighbors(devicePrunedNeighbors &P) {
  chkerr(hipFree(P.newOffset));
  chkerr(hipFree(P.newNeighbors));
  chkerr(hipFree(P.pruneStatus));
}

void freeFlownetwork(deviceFlowNetworkPointers &F) {
  chkerr(hipFree(F.height));
  chkerr(hipFree(F.excess));
  chkerr(hipFree(F.offset));
  chkerr(hipFree(F.neighbors));
  chkerr(hipFree(F.capacity));
  chkerr(hipFree(F.flow));
  chkerr(hipFree(F.flowIndex));
}
